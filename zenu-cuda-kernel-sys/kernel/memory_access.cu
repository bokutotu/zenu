#include "memory_access.h"
#include <hip/hip_runtime.h>

void memory_access_float(float *array, int offset, float *result) {
    hipMemcpy(result, array + offset, sizeof(float), hipMemcpyDeviceToHost);
}

void memory_access_double(double *array, int offset, double *result) {
    hipMemcpy(result, array + offset, sizeof(double), hipMemcpyDeviceToHost);
}

void memory_set_float(float *array, int offset, float value) {
    hipMemcpy(array + offset, &value, sizeof(float), hipMemcpyHostToDevice);
}

void memory_set_double(double *array, int offset, double value) {
    hipMemcpy(array + offset, &value, sizeof(double), hipMemcpyHostToDevice);
}
